#include "hip/hip_runtime.h"
/*************************************************************** -*- C++ -*- ***
 * Copyright (c) 2022 - 2023 NVIDIA Corporation & Affiliates.                  *
 * All rights reserved.                                                        *
 *                                                                             *
 * This source code and the accompanying materials are made available under    *
 * the terms of the Apache License 2.0 which accompanies this distribution.    *
 ******************************************************************************/

#pragma nv_diag_suppress = unsigned_compare_with_zero
#pragma nv_diag_suppress = unrecognized_gcc_pragma

#include "CircuitSimulator.h"
#include "Gates.h"
#include "hip/hip_complex.h"
#include "custatevec.h"
#include <bitset>
#include <complex>
#include <iostream>
#include <random>
#include <set>

namespace {

#define HANDLE_ERROR(x)                                                        \
  {                                                                            \
    const auto err = x;                                                        \
    if (err != CUSTATEVEC_STATUS_SUCCESS) {                                    \
      throw std::runtime_error(fmt::format("[custatevec] %{} in {} (line {})", \
                                           custatevecGetErrorString(err),      \
                                           __FUNCTION__, __LINE__));           \
    }                                                                          \
  };

#define HANDLE_CUDA_ERROR(x)                                                   \
  {                                                                            \
    const auto err = x;                                                        \
    if (err != hipSuccess) {                                                  \
      throw std::runtime_error(fmt::format("[custatevec] %{} in {} (line {})", \
                                           hipGetErrorString(err),            \
                                           __FUNCTION__, __LINE__));           \
    }                                                                          \
  };

/// @brief Initialize the device state vector to the |0...0> state
/// @param sv
/// @param dim
/// @return
template <typename CudaDataType>
__global__ void initializeDeviceStateVector(CudaDataType *sv, int64_t dim) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i == 0) {
    sv[i].x = 1.0;
    sv[i].y = 0.0;
  } else if (i < dim) {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

/// @brief Kernel to set the first N elements of the state vector sv equal to
/// the
// elements provided by the vector sv2. N is the number of elements to set.
// Size of sv must be greater than size of sv2.
/// @param sv
/// @param sv2
/// @param N
/// @return
template <typename T>
__global__ void setFirstNElements(T *sv, const T *__restrict__ sv2, int64_t N) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i < N) {
    sv[i].x = sv2[i].x;
    sv[i].y = sv2[i].y;
  } else {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

/// @brief The CuStateVecCircuitSimulator implements the CircuitSimulator
/// base class to provide a simulator that delegates to the NVIDIA CuStateVec
/// GPU-accelerated library.
template <typename ScalarType = double>
class CuStateVecCircuitSimulator
    : public nvqir::CircuitSimulatorBase<ScalarType> {
protected:
  // This type by default uses FP64
  using DataType = std::complex<ScalarType>;
  using DataVector = std::vector<DataType>;
  using CudaDataType = std::conditional_t<std::is_same_v<ScalarType, float>,
                                          hipFloatComplex, hipDoubleComplex>;

  using nvqir::CircuitSimulatorBase<ScalarType>::tracker;
  using nvqir::CircuitSimulatorBase<ScalarType>::nQubitsAllocated;
  using nvqir::CircuitSimulatorBase<ScalarType>::stateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::calculateStateDim;
  using nvqir::CircuitSimulatorBase<ScalarType>::executionContext;
  using nvqir::CircuitSimulatorBase<ScalarType>::gateToString;
  using nvqir::CircuitSimulatorBase<ScalarType>::x;
  using nvqir::CircuitSimulatorBase<ScalarType>::flushGateQueue;
  using nvqir::CircuitSimulatorBase<ScalarType>::previousStateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::shouldObserveFromSampling;

  /// @brief The statevector that cuStateVec manipulates on the GPU
  void *deviceStateVector = nullptr;

  /// @brief The cuStateVec handle
  custatevecHandle_t handle;

  /// @brief Pointer to potentially needed extra memory
  void *extraWorkspace = nullptr;

  /// @brief The size of the extra workspace
  size_t extraWorkspaceSizeInBytes = 0;

  custatevecComputeType_t cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
  hipDataType cuStateVecCudaDataType = HIP_C_64F;
  std::random_device randomDevice;
  std::mt19937 randomEngine;

  /// @brief Generate a vector of random values
  std::vector<double> randomValues(uint64_t num_samples, double max_value) {
    std::vector<double> rs;
    rs.reserve(num_samples);
    std::uniform_real_distribution<double> distr(0.0, max_value);
    for (uint64_t i = 0; i < num_samples; ++i) {
      rs.emplace_back(distr(randomEngine));
    }
    std::sort(rs.begin(), rs.end());
    return rs;
  }

  /// @brief Convert the pauli rotation gate name to a CUSTATEVEC_PAULI Type
  /// @param type
  /// @return
  custatevecPauli_t pauliStringToEnum(const std::string_view type) {
    if (type == "rx") {
      return CUSTATEVEC_PAULI_X;
    } else if (type == "ry") {
      return CUSTATEVEC_PAULI_Y;
    } else if (type == "rz") {
      return CUSTATEVEC_PAULI_Z;
    }
    printf("Error, should not be here with pauli.\n");
    exit(1);
  }

  /// @brief Apply the matrix to the state vector on the GPU
  /// @param matrix The matrix data as a 1-d array, row-major
  /// @param controls Possible control qubits, can be empty
  /// @param targets Target qubits
  void applyGateMatrix(const DataVector &matrix,
                       const std::vector<int> &controls,
                       const std::vector<int> &targets) {
    HANDLE_ERROR(custatevecApplyMatrixGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nQubitsAllocated, matrix.data(),
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.size(),
        controls.size(), cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    auto localNQubitsAllocated =
        stateDimension > 0 ? std::log2(stateDimension) : 0;

    // apply gate
    HANDLE_ERROR(custatevecApplyMatrix(
        handle, deviceStateVector, cuStateVecCudaDataType,
        localNQubitsAllocated, matrix.data(), cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.data(), targets.size(),
        controls.empty() ? nullptr : controls.data(), nullptr, controls.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
  }

  /// @brief Utility function for applying one-target-qubit rotation operations
  template <typename RotationGateT>
  void oneQubitOneParamApply(const double angle,
                             const std::vector<std::size_t> &controls,
                             const std::size_t qubitIdx) {
    RotationGateT gate;
    std::vector<int> controls32;
    for (auto c : controls)
      controls32.push_back((int)c);
    custatevecPauli_t pauli[] = {pauliStringToEnum(gate.name())};
    int targets[] = {(int)qubitIdx};
    custatevecApplyPauliRotation(handle, deviceStateVector,
                                 cuStateVecCudaDataType, nQubitsAllocated,
                                 -0.5 * angle, pauli, targets, 1,
                                 controls32.data(), nullptr, controls32.size());
  }

  /// @brief Increase the state size by the given number of qubits.
  void addQubitsToState(std::size_t count) override {
    if (count == 0)
      return;

    int dev;
    hipGetDevice(&dev);
    cudaq::info("GPU {} Allocating new qubit array of size {}.", dev, count);

    if (!deviceStateVector) {
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
      HANDLE_ERROR(custatevecCreate(&handle));
    } else {
      // Allocate new state..
      void *newDeviceStateVector;
      HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      setFirstNElements<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(newDeviceStateVector),
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          previousStateDimension);
      hipFree(deviceStateVector);
      deviceStateVector = newDeviceStateVector;
    }
  }

  /// @brief Increase the state size by one qubit.
  void addQubitToState() override {
    // Update the state vector
    if (!deviceStateVector) {
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
      HANDLE_ERROR(custatevecCreate(&handle));
    } else {
      // Allocate new state..
      void *newDeviceStateVector;
      HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      setFirstNElements<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(newDeviceStateVector),
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          previousStateDimension);
      hipFree(deviceStateVector);
      deviceStateVector = newDeviceStateVector;
    }
  }

  /// @brief Reset the qubit state.
  void deallocateStateImpl() override {
    HANDLE_ERROR(custatevecDestroy(handle));
    HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
    if (extraWorkspaceSizeInBytes)
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
    deviceStateVector = nullptr;
    extraWorkspaceSizeInBytes = 0;
  }

  /// @brief Apply the given GateApplicationTask
  void applyGate(const typename nvqir::CircuitSimulatorBase<
                 ScalarType>::GateApplicationTask &task) override {
    std::vector<int> controls, targets;
    std::transform(task.controls.begin(), task.controls.end(),
                   std::back_inserter(controls),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    std::transform(task.targets.begin(), task.targets.end(),
                   std::back_inserter(targets),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    // If we have no parameters, just apply the matrix.
    if (task.parameters.empty()) {
      applyGateMatrix(task.matrix, controls, targets);
      return;
    }

    // If we have parameters, it may be more efficient to
    // compute with custatevecApplyPauliRotation
    if (task.operationName == "rx") {
      oneQubitOneParamApply<nvqir::rx<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "ry") {
      oneQubitOneParamApply<nvqir::ry<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "rz") {
      oneQubitOneParamApply<nvqir::rz<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else {
      // Fallback to just applying the gate.
      applyGateMatrix(task.matrix, controls, targets);
    }
  }

  /// @brief Set the state back to the |0> state on the
  /// current number of qubits
  void setToZeroState() override {
    constexpr int32_t threads_per_block = 256;
    uint32_t n_blocks =
        (stateDimension + threads_per_block - 1) / threads_per_block;
    initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
        reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
  }

public:
  /// @brief The constructor
  CuStateVecCircuitSimulator() {
    if constexpr (std::is_same_v<ScalarType, float>) {
      cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
      cuStateVecCudaDataType = HIP_C_32F;
    }

    hipFree(0);
    randomEngine = std::mt19937(randomDevice());
  }

  /// The destructor
  virtual ~CuStateVecCircuitSimulator() = default;

  void setRandomSeed(std::size_t randomSeed) override {
    randomEngine = std::mt19937(randomSeed);
  }

  /// @brief Measure operation
  /// @param qubitIdx
  /// @return
  bool measureQubit(const std::size_t qubitIdx) override {
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    cudaq::info("Measured qubit {} -> {}", qubitIdx, parity);
    return parity == 1 ? true : false;
  }

  /// @brief Reset the qubit
  /// @param qubitIdx
  void resetQubit(const std::size_t qubitIdx) override {
    flushGateQueue();
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    if (parity) {
      x(qubitIdx);
    }
  }

  /// @brief Override base class functionality for a general Pauli
  /// rotation to delegate to the performant custatevecApplyPauliRotation.
  void applyExpPauli(double theta, const std::vector<std::size_t> &controlIds,
                     const std::vector<std::size_t> &qubits,
                     const cudaq::spin_op &op) override {
    flushGateQueue();
    cudaq::info(" [cusv decomposing] exp_pauli({}, {})", theta,
                op.to_string(false));
    std::vector<int> controls, targets;
    for (const auto &bit : controlIds)
      controls.emplace_back(static_cast<int>(bit));
    std::vector<custatevecPauli_t> paulis;
    op.for_each_pauli([&](cudaq::pauli p, std::size_t i) {
      if (p == cudaq::pauli::I)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_I);
      else if (p == cudaq::pauli::X)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_X);
      else if (p == cudaq::pauli::Y)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Y);
      else
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Z);

      targets.push_back(qubits[i]);
    });

    HANDLE_ERROR(custatevecApplyPauliRotation(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        theta, paulis.data(), targets.data(), targets.size(), controls.data(),
        nullptr, controls.size()));
  }

  /// @brief Compute the operator expectation value, with respect to
  /// the current state vector, directly on GPU with the
  /// given the operator matrix and target qubit indices.
  auto getExpectationFromOperatorMatrix(const std::complex<double> *matrix,
                                        const std::vector<std::size_t> &tgts) {
    void *extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // Convert the size_t tgts into ints
    std::vector<int> tgtsInt(tgts.size());
    std::transform(tgts.begin(), tgts.end(), tgtsInt.begin(),
                   [&](std::size_t x) { return static_cast<int>(x); });
    // our bit ordering is reversed.
    std::reverse(tgtsInt.begin(), tgtsInt.end());
    size_t nIndexBits = nQubitsAllocated;

    // check the size of external workspace
    HANDLE_ERROR(custatevecComputeExpectationGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nIndexBits, matrix,
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, tgts.size(),
        cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0) {
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));
    }

    double expect;

    // compute expectation
    HANDLE_ERROR(custatevecComputeExpectation(
        handle, deviceStateVector, cuStateVecCudaDataType, nIndexBits, &expect,
        HIP_R_64F, nullptr, matrix, cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, tgtsInt.data(), tgts.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
    if (extraWorkspaceSizeInBytes)
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));

    return expect;
  }

  /// @brief We can compute Observe from the matrix for a
  /// reasonable number of qubits, otherwise we should compute it
  /// via sampling
  bool canHandleObserve() override {
    // Do not compute <H> from matrix if shots based sampling requested
    if (executionContext &&
        executionContext->shots != static_cast<std::size_t>(-1)) {
      return false;
    }

    /// Seems that FP32 is faster with
    /// custatevecComputeExpectationsOnPauliBasis
    if constexpr (std::is_same_v<ScalarType, float>) {
      return false;
    }

    return !shouldObserveFromSampling();
  }

  /// @brief Compute the expected value from the observable matrix.
  cudaq::ExecutionResult observe(const cudaq::spin_op &op) override {

    flushGateQueue();

    // The op is on the following target bits.
    std::set<std::size_t> targets;
    op.for_each_term([&](cudaq::spin_op &term) {
      term.for_each_pauli(
          [&](cudaq::pauli p, std::size_t idx) { targets.insert(idx); });
    });

    std::vector<std::size_t> targetsVec(targets.begin(), targets.end());

    // Get the matrix
    auto matrix = op.to_matrix();
    /// Compute the expectation value.
    auto ee = getExpectationFromOperatorMatrix(matrix.data(), targetsVec);
    return cudaq::ExecutionResult({}, ee);
  }

  /// @brief Sample the multi-qubit state.
  cudaq::ExecutionResult sample(const std::vector<std::size_t> &measuredBits,
                                const int shots) override {
    double expVal = 0.0;
    // cudaq::CountsDictionary counts;
    std::vector<custatevecPauli_t> z_pauli;
    std::vector<int> measuredBits32;
    for (auto m : measuredBits) {
      measuredBits32.push_back(m);
      z_pauli.push_back(CUSTATEVEC_PAULI_Z);
    }

    if (shots < 1) {
      // Just compute the expected value on <Z...Z>
      const uint32_t nBasisBitsArray[] = {(uint32_t)measuredBits.size()};
      const int *basisBitsArray[] = {measuredBits32.data()};
      const custatevecPauli_t *pauliArray[] = {z_pauli.data()};
      double expectationValues[1];
      HANDLE_ERROR(custatevecComputeExpectationsOnPauliBasis(
          handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
          expectationValues, pauliArray, 1, basisBitsArray, nBasisBitsArray));
      expVal = expectationValues[0];
      cudaq::info("Computed expectation value = {}", expVal);
      return cudaq::ExecutionResult{expVal};
    }

    // Grab some random seed values and create the sampler
    auto randomValues_ = randomValues(shots, 1.0);
    custatevecSamplerDescriptor_t sampler;
    HANDLE_ERROR(custatevecSamplerCreate(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &sampler, shots, &extraWorkspaceSizeInBytes));
    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0) {
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));
    }

    // Run the sampling preprocess step.
    HANDLE_ERROR(custatevecSamplerPreprocess(handle, sampler, extraWorkspace,
                                             extraWorkspaceSizeInBytes));

    // Sample!
    custatevecIndex_t bitstrings0[shots];
    HANDLE_ERROR(custatevecSamplerSample(
        handle, sampler, bitstrings0, measuredBits32.data(),
        measuredBits32.size(), randomValues_.data(), shots,
        CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

    std::vector<std::string> sequentialData;

    cudaq::ExecutionResult counts;

    // We've sampled, convert the results to our ExecutionResult counts
    for (int i = 0; i < shots; ++i) {
      auto bitstring = std::bitset<64>(bitstrings0[i])
                           .to_string()
                           .erase(0, 64 - measuredBits.size());
      std::reverse(bitstring.begin(), bitstring.end());
      sequentialData.push_back(bitstring);
      counts.appendResult(bitstring, 1);
    }

    // Compute the expectation value from the counts
    for (auto &kv : counts.counts) {
      auto par = cudaq::sample_result::has_even_parity(kv.first);
      auto p = kv.second / (double)shots;
      if (!par) {
        p = -p;
      }
      expVal += p;
    }

    counts.expectationValue = expVal;
    return counts;
  }

  cudaq::State getStateData() override {
    // Handle empty state (e.g., no qubit allocation)
    if (stateDimension == 0)
      return cudaq::State{{stateDimension}, {}};

    std::vector<std::complex<ScalarType>> tmp(stateDimension);
    // Use custatevec accessor to retrieve the view
    custatevecAccessorDescriptor_t accessor;
    const uint32_t nIndexBits = std::log2(stateDimension);
    // Note: we use MSB bit ordering when reporting the state vector
    // hence, bit ordering vector = [N-1, N-2, ..., 0]
    std::vector<int32_t> bitOrdering(nIndexBits);
    std::iota(std::rbegin(bitOrdering), std::rend(bitOrdering), 0);
    std::size_t extraWorkspaceSizeInBytes = 0;
    // create accessor view
    HANDLE_ERROR(custatevecAccessorCreateView(
        handle, deviceStateVector, cuStateVecCudaDataType, nIndexBits,
        &accessor, bitOrdering.data(), bitOrdering.size(),
        /*maskBitString*/ nullptr, /*maskOrdering*/ nullptr,
        /*maskLen*/ 0, &extraWorkspaceSizeInBytes));
    // allocate external workspace if necessary
    void *extraWorkspace = nullptr;
    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    // set external workspace
    HANDLE_ERROR(custatevecAccessorSetExtraWorkspace(
        handle, accessor, extraWorkspace, extraWorkspaceSizeInBytes));

    // get all state vector components: [0, stateDimension)
    HANDLE_ERROR(custatevecAccessorGet(handle, accessor, tmp.data(),
                                       /*begin*/ 0,
                                       /*end*/
                                       stateDimension));
    // destroy descriptor
    HANDLE_ERROR(custatevecAccessorDestroy(accessor));
    // free extra workspace if allocated
    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));

    if constexpr (std::is_same_v<ScalarType, float>) {
      std::vector<std::complex<double>> data;
      std::transform(tmp.begin(), tmp.end(), std::back_inserter(data),
                     [](std::complex<float> &el) -> std::complex<double> {
                       return {static_cast<double>(el.real()),
                               static_cast<double>(el.imag())};
                     });
      return cudaq::State{{stateDimension}, data};
    } else {
      return cudaq::State{{stateDimension}, tmp};
    }
  }

  std::string name() const override;
  NVQIR_SIMULATOR_CLONE_IMPL(CuStateVecCircuitSimulator<ScalarType>)
};
} // namespace

#ifndef __NVQIR_CUSTATEVEC_TOGGLE_CREATE
template <>
std::string CuStateVecCircuitSimulator<double>::name() const {
  return "custatevec-fp64";
}
/// Register this Simulator with NVQIR.
NVQIR_REGISTER_SIMULATOR(CuStateVecCircuitSimulator<>, custatevec_fp64)
#endif
