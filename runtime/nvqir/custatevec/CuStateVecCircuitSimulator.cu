#include "hip/hip_runtime.h"
/*************************************************************** -*- C++ -*- ***
 * Copyright (c) 2022 - 2024 NVIDIA Corporation & Affiliates.                  *
 * All rights reserved.                                                        *
 *                                                                             *
 * This source code and the accompanying materials are made available under    *
 * the terms of the Apache License 2.0 which accompanies this distribution.    *
 ******************************************************************************/

#pragma nv_diag_suppress = unsigned_compare_with_zero
#pragma nv_diag_suppress = unrecognized_gcc_pragma

#include "CircuitSimulator.h"
#include "CuStateVecState.h"
#include "Gates.h"
#include "Timing.h"
#include "hip/hip_complex.h"
#include "custatevec.h"
#include <bitset>
#include <complex>
#include <iostream>
#include <random>
#include <set>

namespace {

/// @brief Initialize the device state vector to the |0...0> state
/// @param sv
/// @param dim
/// @return
template <typename CudaDataType>
__global__ void initializeDeviceStateVector(CudaDataType *sv, int64_t dim) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i == 0) {
    sv[i].x = 1.0;
    sv[i].y = 0.0;
  } else if (i < dim) {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

/// @brief Kernel to set the first N elements of the state vector sv equal to
/// the
// elements provided by the vector sv2. N is the number of elements to set.
// Size of sv must be greater than size of sv2.
/// @param sv
/// @param sv2
/// @param N
/// @return
template <typename T>
__global__ void setFirstNElements(T *sv, const T *__restrict__ sv2, int64_t N) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i < N) {
    sv[i].x = sv2[i].x;
    sv[i].y = sv2[i].y;
  } else {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

// kronprod functions adapted from
// https://github.com/DmitryLyakh/TAL_SH/blob/3cefc2133a68b67c515f4b68a0ed9e3c66e4b4b2/tensor_algebra_gpu_nvidia.cu#L745

#define THRDS_ARRAY_PRODUCT 256

#pragma push
#pragma nv_diag_suppress 177
__device__ __host__ hipDoubleComplex operator*(hipDoubleComplex a,
                                              hipDoubleComplex b) {
  return hipCmul(a, b);
}
__device__ __host__ hipDoubleComplex operator+(hipDoubleComplex a,
                                              hipDoubleComplex b) {
  return hipCadd(a, b);
}
__device__ __host__ hipFloatComplex operator*(hipFloatComplex a,
                                             hipFloatComplex b) {
  return hipCmulf(a, b);
}
__device__ __host__ hipFloatComplex operator+(hipFloatComplex a,
                                             hipFloatComplex b) {
  return hipCaddf(a, b);
}

template <typename T>
__global__ void kronprod(size_t tsize1, const T *arr1, size_t tsize2,
                         const T *arr2, T *arr0) {
  __shared__ T lbuf[THRDS_ARRAY_PRODUCT + 1], rbuf[THRDS_ARRAY_PRODUCT];
  size_t _ib, _in, _jb, _jn, _tx, _jc, _ja;

  _tx = (size_t)threadIdx.x;
  for (_jb = blockIdx.y * THRDS_ARRAY_PRODUCT; _jb < tsize2;
       _jb += gridDim.y * THRDS_ARRAY_PRODUCT) {
    if (_jb + THRDS_ARRAY_PRODUCT > tsize2) {
      _jn = tsize2 - _jb;
    } else {
      _jn = THRDS_ARRAY_PRODUCT;
    }

    if (_tx < _jn)
      rbuf[_tx] = arr2[_jb + _tx];

    for (_ib = blockIdx.x * THRDS_ARRAY_PRODUCT; _ib < tsize1;
         _ib += gridDim.x * THRDS_ARRAY_PRODUCT) {
      if (_ib + THRDS_ARRAY_PRODUCT > tsize1) {
        _in = tsize1 - _ib;
      } else {
        _in = THRDS_ARRAY_PRODUCT;
      }

      if (_tx < _in)
        lbuf[_tx] = arr1[_ib + _tx];

      __syncthreads();
      for (_jc = 0; _jc < _jn; _jc++) {
        if (_tx < _in) {
          _ja = (_jb + _jc) * tsize1 + (_ib + _tx);
          arr0[_ja] = arr0[_ja] + lbuf[_tx] * rbuf[_jc];
        }
      }
      __syncthreads();
    }
  }
  return;
}
#pragma pop

/// @brief The CuStateVecCircuitSimulator implements the CircuitSimulator
/// base class to provide a simulator that delegates to the NVIDIA CuStateVec
/// GPU-accelerated library.
template <typename ScalarType = double>
class CuStateVecCircuitSimulator
    : public nvqir::CircuitSimulatorBase<ScalarType> {
protected:
  // This type by default uses FP64
  using DataType = std::complex<ScalarType>;
  using DataVector = std::vector<DataType>;
  using CudaDataType = std::conditional_t<std::is_same_v<ScalarType, float>,
                                          hipFloatComplex, hipDoubleComplex>;

  using nvqir::CircuitSimulatorBase<ScalarType>::tracker;
  using nvqir::CircuitSimulatorBase<ScalarType>::nQubitsAllocated;
  using nvqir::CircuitSimulatorBase<ScalarType>::stateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::calculateStateDim;
  using nvqir::CircuitSimulatorBase<ScalarType>::executionContext;
  using nvqir::CircuitSimulatorBase<ScalarType>::gateToString;
  using nvqir::CircuitSimulatorBase<ScalarType>::x;
  using nvqir::CircuitSimulatorBase<ScalarType>::flushGateQueue;
  using nvqir::CircuitSimulatorBase<ScalarType>::previousStateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::shouldObserveFromSampling;
  using nvqir::CircuitSimulatorBase<ScalarType>::summaryData;

  /// @brief The statevector that cuStateVec manipulates on the GPU
  void *deviceStateVector = nullptr;

  /// @brief The cuStateVec handle
  custatevecHandle_t handle = nullptr;

  /// @brief Pointer to potentially needed extra memory
  void *extraWorkspace = nullptr;

  /// @brief The size of the extra workspace
  size_t extraWorkspaceSizeInBytes = 0;

  custatevecComputeType_t cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
  hipDataType cuStateVecCudaDataType = HIP_C_64F;
  std::random_device randomDevice;
  std::mt19937 randomEngine;
  bool ownsDeviceVector = true;

  /// @brief Generate a vector of random values
  std::vector<double> randomValues(uint64_t num_samples, double max_value) {
    std::vector<double> rs;
    rs.reserve(num_samples);
    std::uniform_real_distribution<double> distr(0.0, max_value);
    for (uint64_t i = 0; i < num_samples; ++i) {
      rs.emplace_back(distr(randomEngine));
    }
    std::sort(rs.begin(), rs.end());
    return rs;
  }

  /// @brief Convert the pauli rotation gate name to a CUSTATEVEC_PAULI Type
  /// @param type
  /// @return
  custatevecPauli_t pauliStringToEnum(const std::string_view type) {
    if (type == "rx") {
      return CUSTATEVEC_PAULI_X;
    } else if (type == "ry") {
      return CUSTATEVEC_PAULI_Y;
    } else if (type == "rz") {
      return CUSTATEVEC_PAULI_Z;
    }
    printf("Error, should not be here with pauli.\n");
    exit(1);
  }

  /// @brief Apply the matrix to the state vector on the GPU
  /// @param matrix The matrix data as a 1-d array, row-major
  /// @param controls Possible control qubits, can be empty
  /// @param targets Target qubits
  void applyGateMatrix(const DataVector &matrix,
                       const std::vector<int> &controls,
                       const std::vector<int> &targets) {
    HANDLE_ERROR(custatevecApplyMatrixGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nQubitsAllocated, matrix.data(),
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.size(),
        controls.size(), cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    auto localNQubitsAllocated =
        stateDimension > 0 ? std::log2(stateDimension) : 0;

    // apply gate
    HANDLE_ERROR(custatevecApplyMatrix(
        handle, deviceStateVector, cuStateVecCudaDataType,
        localNQubitsAllocated, matrix.data(), cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.data(), targets.size(),
        controls.empty() ? nullptr : controls.data(), nullptr, controls.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));

    if (extraWorkspace) {
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
      extraWorkspace = nullptr;
    }
  }

  /// @brief Utility function for applying one-target-qubit rotation operations
  template <typename RotationGateT>
  void oneQubitOneParamApply(const double angle,
                             const std::vector<std::size_t> &controls,
                             const std::size_t qubitIdx) {
    RotationGateT gate;
    std::vector<int> controls32;
    for (auto c : controls)
      controls32.push_back((int)c);
    custatevecPauli_t pauli[] = {pauliStringToEnum(gate.name())};
    int targets[] = {(int)qubitIdx};
    HANDLE_ERROR(custatevecApplyPauliRotation(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        -0.5 * angle, pauli, targets, 1, controls32.data(), nullptr,
        controls32.size()));
  }

  /// @brief Nice utility function to have to print the state vector contents on
  /// GPU.
  void printStateFromGPU(const std::string &name, void *ptr, std::size_t size) {
    std::vector<std::complex<ScalarType>> tmp(size);
    hipMemcpy(tmp.data(), ptr, size * sizeof(std::complex<ScalarType>),
               hipMemcpyDeviceToHost);
    for (auto &r : tmp)
      printf("%s: (%.12lf, %.12lf)\n", name.c_str(), r.real(), r.imag());
    printf("\n");
  }

  /// @brief Increase the state size by the given number of qubits.
  void addQubitsToState(std::size_t count, const void *stateIn) override {
    ScopedTraceWithContext("CuStateVecCircuitSimulator::addQubitsToState",
                           count);
    if (count == 0)
      return;

    // Cast the state, at this point an error would
    // have been thrown if it is not of the right floating point type
    std::complex<ScalarType> *state =
        reinterpret_cast<std::complex<ScalarType> *>(
            const_cast<void *>(stateIn));

    int dev;
    HANDLE_CUDA_ERROR(hipGetDevice(&dev));
    cudaq::info("GPU {} Allocating new qubit array of size {}.", dev, count);

    constexpr int32_t threads_per_block = 256;
    uint32_t n_blocks =
        (stateDimension + threads_per_block - 1) / threads_per_block;

    // Check if this is the first time to allocate, if so
    // the allocation is much easier
    if (!deviceStateVector) {
      // Create the memory and the handle
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      HANDLE_ERROR(custatevecCreate(&handle));
      ownsDeviceVector = true;
      // If no state provided, initialize to the zero state
      if (state == nullptr) {
        initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
            reinterpret_cast<CudaDataType *>(deviceStateVector),
            stateDimension);
        return;
      }

      // User state provided...

      // FIXME handle case where pointer is a device pointer

      // First allocation, so just set the user provided data here
      ScopedTraceWithContext(
          "CuStateVecCircuitSimulator::addQubitsToState hipMemcpy",
          stateDimension * sizeof(CudaDataType));
      HANDLE_CUDA_ERROR(hipMemcpy(deviceStateVector, state,
                                   stateDimension * sizeof(CudaDataType),
                                   hipMemcpyHostToDevice));
      return;
    }

    // State already exists, need to allocate new state and compute
    // kronecker product with existing state

    // Allocate new vector to place the kron prod result
    void *newDeviceStateVector;
    HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                 stateDimension * sizeof(CudaDataType)));

    // Place the state data on device. Could be that
    // we just need the zero state, or the user could have provided one
    void *otherState;
    HANDLE_CUDA_ERROR(hipMalloc((void **)&otherState,
                                 (1UL << count) * sizeof(CudaDataType)));
    if (state == nullptr) {
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(otherState), (1UL << count));
    } else {

      // FIXME Handle case where data is already on GPU
      HANDLE_CUDA_ERROR(hipMemcpy(otherState, state,
                                   (1UL << count) * sizeof(CudaDataType),
                                   hipMemcpyHostToDevice));
    }

    {
      ScopedTraceWithContext(
          "CuStateVecCircuitSimulator::addQubitsToState kronprod");
      // Compute the kronecker product
      kronprod<CudaDataType><<<n_blocks, threads_per_block>>>(
          previousStateDimension,
          reinterpret_cast<CudaDataType *>(deviceStateVector), (1UL << count),
          reinterpret_cast<CudaDataType *>(otherState),
          reinterpret_cast<CudaDataType *>(newDeviceStateVector));
      HANDLE_CUDA_ERROR(hipGetLastError());
    }
    // Free the old vectors we don't need anymore.
    HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
    HANDLE_CUDA_ERROR(hipFree(otherState));
    deviceStateVector = newDeviceStateVector;
  }

  void addQubitsToState(const cudaq::SimulationState &in_state) override {
    const cudaq::CusvState<ScalarType> *const casted =
        dynamic_cast<const cudaq::CusvState<ScalarType> *>(&in_state);
    if (!casted)
      throw std::invalid_argument(
          "[CuStateVecCircuitSimulator] Incompatible state input");

    if (!deviceStateVector) {
      // Create the memory and the handle
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      ownsDeviceVector = true;
      HANDLE_ERROR(custatevecCreate(&handle));
      ScopedTraceWithContext(
          "CuStateVecCircuitSimulator::addQubitsToState hipMemcpy");
      // First allocation, so just copy the user provided data (device mem) here
      HANDLE_CUDA_ERROR(hipMemcpy(
          deviceStateVector, casted->getDevicePointer(),
          stateDimension * sizeof(CudaDataType), hipMemcpyDeviceToDevice));
      return;
    }

    // Expanding the state
    // Allocate new vector to place the kron prod result
    void *newDeviceStateVector;
    HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                 stateDimension * sizeof(CudaDataType)));
    constexpr int32_t threads_per_block = 256;
    uint32_t n_blocks =
        (stateDimension + threads_per_block - 1) / threads_per_block;
    {
      ScopedTraceWithContext(
          "CuStateVecCircuitSimulator::addQubitsToState kronprod");
      // Compute the kronecker product
      kronprod<CudaDataType><<<n_blocks, threads_per_block>>>(
          previousStateDimension,
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          (1UL << in_state.getNumQubits()),
          reinterpret_cast<const CudaDataType *>(casted->getDevicePointer()),
          reinterpret_cast<CudaDataType *>(newDeviceStateVector));
      HANDLE_CUDA_ERROR(hipGetLastError());
    }
    // Free the old state we don't need anymore.
    // Note: the devicePtr of the input state is owned by the caller.
    HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
    deviceStateVector = newDeviceStateVector;
  }

  /// @brief Increase the state size by one qubit.
  void addQubitToState() override {
    ScopedTraceWithContext("CuStateVecCircuitSimulator::addQubitToState");
    // Update the state vector
    if (!deviceStateVector) {
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
      HANDLE_ERROR(custatevecCreate(&handle));
    } else {
      // Allocate new state..
      void *newDeviceStateVector;
      HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      setFirstNElements<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(newDeviceStateVector),
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          previousStateDimension);
      HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
      deviceStateVector = newDeviceStateVector;
    }
  }

  /// @brief Reset the qubit state.
  void deallocateStateImpl() override {
    if (deviceStateVector)
      HANDLE_ERROR(custatevecDestroy(handle));
    if (deviceStateVector && ownsDeviceVector) {
      HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
    }
    if (extraWorkspace) {
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
      extraWorkspace = nullptr;
    }
    deviceStateVector = nullptr;
    extraWorkspaceSizeInBytes = 0;
  }

  /// @brief Apply the given GateApplicationTask
  void applyGate(const typename nvqir::CircuitSimulatorBase<
                 ScalarType>::GateApplicationTask &task) override {
    std::vector<int> controls, targets;
    std::transform(task.controls.begin(), task.controls.end(),
                   std::back_inserter(controls),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    std::transform(task.targets.begin(), task.targets.end(),
                   std::back_inserter(targets),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    // If we have no parameters, just apply the matrix.
    if (task.parameters.empty()) {
      applyGateMatrix(task.matrix, controls, targets);
      return;
    }

    // If we have parameters, it may be more efficient to
    // compute with custatevecApplyPauliRotation
    if (task.operationName == "rx") {
      oneQubitOneParamApply<nvqir::rx<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "ry") {
      oneQubitOneParamApply<nvqir::ry<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "rz") {
      oneQubitOneParamApply<nvqir::rz<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else {
      // Fallback to just applying the gate.
      applyGateMatrix(task.matrix, controls, targets);
    }
  }

  /// @brief Set the state back to the |0> state on the
  /// current number of qubits
  void setToZeroState() override {
    constexpr int32_t threads_per_block = 256;
    uint32_t n_blocks =
        (stateDimension + threads_per_block - 1) / threads_per_block;
    initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
        reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
  }

public:
  /// @brief The constructor
  CuStateVecCircuitSimulator() {
    if constexpr (std::is_same_v<ScalarType, float>) {
      cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
      cuStateVecCudaDataType = HIP_C_32F;
    }

    // Populate the correct name so it is printed correctly during
    // deconstructor.
    summaryData.name = name();

    HANDLE_CUDA_ERROR(hipFree(0));
    randomEngine = std::mt19937(randomDevice());
  }

  /// The destructor
  virtual ~CuStateVecCircuitSimulator() = default;

  void setRandomSeed(std::size_t randomSeed) override {
    randomEngine = std::mt19937(randomSeed);
  }

  /// @brief Device synchronization
  void synchronize() override { HANDLE_CUDA_ERROR(hipDeviceSynchronize()); }

  /// @brief Measure operation
  /// @param qubitIdx
  /// @return
  bool measureQubit(const std::size_t qubitIdx) override {
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    cudaq::info("Measured qubit {} -> {}", qubitIdx, parity);
    return parity == 1 ? true : false;
  }

  /// @brief Reset the qubit
  /// @param qubitIdx
  void resetQubit(const std::size_t qubitIdx) override {
    flushGateQueue();
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    if (parity) {
      x(qubitIdx);
    }
  }

  /// @brief Override base class functionality for a general Pauli
  /// rotation to delegate to the performant custatevecApplyPauliRotation.
  void applyExpPauli(double theta, const std::vector<std::size_t> &controlIds,
                     const std::vector<std::size_t> &qubits,
                     const cudaq::spin_op &op) override {
    flushGateQueue();
    cudaq::info(" [cusv decomposing] exp_pauli({}, {})", theta,
                op.to_string(false));
    std::vector<int> controls, targets;
    for (const auto &bit : controlIds)
      controls.emplace_back(static_cast<int>(bit));
    std::vector<custatevecPauli_t> paulis;
    op.for_each_pauli([&](cudaq::pauli p, std::size_t i) {
      if (p == cudaq::pauli::I)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_I);
      else if (p == cudaq::pauli::X)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_X);
      else if (p == cudaq::pauli::Y)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Y);
      else
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Z);

      targets.push_back(qubits[i]);
    });

    HANDLE_ERROR(custatevecApplyPauliRotation(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        theta, paulis.data(), targets.data(), targets.size(), controls.data(),
        nullptr, controls.size()));
  }

  /// @brief Compute the operator expectation value, with respect to
  /// the current state vector, directly on GPU with the
  /// given the operator matrix and target qubit indices.
  auto getExpectationFromOperatorMatrix(const std::complex<double> *matrix,
                                        const std::vector<std::size_t> &tgts) {
    // Convert the size_t tgts into ints
    std::vector<int> tgtsInt(tgts.size());
    std::transform(tgts.begin(), tgts.end(), tgtsInt.begin(),
                   [&](std::size_t x) { return static_cast<int>(x); });
    // our bit ordering is reversed.
    size_t nIndexBits = nQubitsAllocated;

    // check the size of external workspace
    HANDLE_ERROR(custatevecComputeExpectationGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nIndexBits, matrix,
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, tgts.size(),
        cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    double expect;

    // compute expectation
    HANDLE_ERROR(custatevecComputeExpectation(
        handle, deviceStateVector, cuStateVecCudaDataType, nIndexBits, &expect,
        HIP_R_64F, nullptr, matrix, cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, tgtsInt.data(), tgts.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));

    if (extraWorkspace) {
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
      extraWorkspace = nullptr;
    }

    return expect;
  }

  /// @brief We can compute Observe from the matrix for a
  /// reasonable number of qubits, otherwise we should compute it
  /// via sampling
  bool canHandleObserve() override {
    // Do not compute <H> from matrix if shots based sampling requested
    // i.e., a valid shots count value was set.
    // Note: -1 is also used to denote non-sampling execution. Hence, we need to
    // check for this particular -1 value as being casted to an unsigned type.
    if (executionContext && executionContext->shots > 0 &&
        executionContext->shots != static_cast<std::size_t>(-1)) {
      return false;
    }

    // If no shots requested (exact expectation calulation), don't use
    // term-by-term observe as the default since
    // `CuStateVecCircuitSimulator::observe` will do a batched expectation value
    // calculation to compute all expectation values for all terms at once.
    return !shouldObserveFromSampling(/*defaultConfig=*/false);
  }

  /// @brief Compute the expected value from the observable matrix.
  cudaq::observe_result observe(const cudaq::spin_op &op) override {
    // Use batched custatevecComputeExpectationsOnPauliBasis to compute all term
    // expectation values in one go
    uint32_t nPauliOperatorArrays = op.num_terms();

    // custatevecComputeExpectationsOnPauliBasis will throw errors if
    // nPauliOperatorArrays is 0, so catch that case early.
    if (nPauliOperatorArrays == 0)
      return cudaq::observe_result{};

    // Stable holders of vectors since we need to send vectors of pointers to
    // custatevec
    std::deque<std::vector<custatevecPauli_t>> pauliOperatorsArrayHolder;
    std::deque<std::vector<int32_t>> basisBitsArrayHolder;
    std::vector<const custatevecPauli_t *> pauliOperatorsArray;
    std::vector<const int32_t *> basisBitsArray;
    std::vector<std::complex<double>> coeffs;
    std::vector<uint32_t> nBasisBitsArray;
    pauliOperatorsArray.reserve(nPauliOperatorArrays);
    basisBitsArray.reserve(nPauliOperatorArrays);
    coeffs.reserve(nPauliOperatorArrays);
    nBasisBitsArray.reserve(nPauliOperatorArrays);
    // Helper to convert Pauli enums
    const auto cudaqToCustateVec = [](cudaq::pauli pauli) -> custatevecPauli_t {
      switch (pauli) {
      case cudaq::pauli::I:
        return CUSTATEVEC_PAULI_I;
      case cudaq::pauli::X:
        return CUSTATEVEC_PAULI_X;
      case cudaq::pauli::Y:
        return CUSTATEVEC_PAULI_Y;
      case cudaq::pauli::Z:
        return CUSTATEVEC_PAULI_Z;
      }
      __builtin_unreachable();
    };

    // Contruct data to send on to custatevec
    std::vector<std::string> termStrs;
    termStrs.reserve(nPauliOperatorArrays);
    op.for_each_term([&](cudaq::spin_op &term) {
      coeffs.emplace_back(term.get_coefficient());
      std::vector<custatevecPauli_t> paulis;
      std::vector<int32_t> idxs;
      paulis.reserve(term.num_qubits());
      idxs.reserve(term.num_qubits());
      term.for_each_pauli([&](cudaq::pauli p, std::size_t idx) {
        if (p != cudaq::pauli::I) {
          paulis.emplace_back(cudaqToCustateVec(p));
          idxs.emplace_back(idx);
          // Only X and Y pauli's translate to applied gates
          if (p != cudaq::pauli::Z) {
            // One operation for applying the term
            summaryData.svGateUpdate(/*nControls=*/0, /*nTargets=*/1,
                                     stateDimension,
                                     stateDimension * sizeof(DataType));
            // And one operation for un-applying the term
            summaryData.svGateUpdate(/*nControls=*/0, /*nTargets=*/1,
                                     stateDimension,
                                     stateDimension * sizeof(DataType));
          }
        }
      });
      pauliOperatorsArrayHolder.emplace_back(std::move(paulis));
      basisBitsArrayHolder.emplace_back(std::move(idxs));
      pauliOperatorsArray.emplace_back(pauliOperatorsArrayHolder.back().data());
      basisBitsArray.emplace_back(basisBitsArrayHolder.back().data());
      nBasisBitsArray.emplace_back(pauliOperatorsArrayHolder.back().size());
      termStrs.emplace_back(term.to_string(false));
    });
    std::vector<double> expectationValues(nPauliOperatorArrays);
    HANDLE_ERROR(custatevecComputeExpectationsOnPauliBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        expectationValues.data(), pauliOperatorsArray.data(),
        nPauliOperatorArrays, basisBitsArray.data(), nBasisBitsArray.data()));
    std::complex<double> expVal = 0.0;
    std::vector<cudaq::ExecutionResult> results;
    results.reserve(nPauliOperatorArrays);
    for (uint32_t i = 0; i < nPauliOperatorArrays; ++i) {
      expVal += coeffs[i] * expectationValues[i];
      results.emplace_back(
          cudaq::ExecutionResult({}, termStrs[i], expectationValues[i]));
    }
    cudaq::sample_result perTermData(static_cast<double>(expVal.real()),
                                     results);
    return cudaq::observe_result(static_cast<double>(expVal.real()), op,
                                 perTermData);
  }

  /// @brief Sample the multi-qubit state.
  cudaq::ExecutionResult sample(const std::vector<std::size_t> &measuredBits,
                                const int shots) override {
    ScopedTraceWithContext(cudaq::TIMING_SAMPLE, "CuStateVecSimulator::sample");
    double expVal = 0.0;
    // cudaq::CountsDictionary counts;
    std::vector<custatevecPauli_t> z_pauli;
    std::vector<int> measuredBits32;
    for (auto m : measuredBits) {
      measuredBits32.push_back(m);
      z_pauli.push_back(CUSTATEVEC_PAULI_Z);
    }

    if (shots < 1) {
      // Just compute the expected value on <Z...Z>
      const uint32_t nBasisBitsArray[] = {(uint32_t)measuredBits.size()};
      const int *basisBitsArray[] = {measuredBits32.data()};
      const custatevecPauli_t *pauliArray[] = {z_pauli.data()};
      double expectationValues[1];
      HANDLE_ERROR(custatevecComputeExpectationsOnPauliBasis(
          handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
          expectationValues, pauliArray, 1, basisBitsArray, nBasisBitsArray));
      expVal = expectationValues[0];
      cudaq::info("Computed expectation value = {}", expVal);
      return cudaq::ExecutionResult{expVal};
    }

    // Grab some random seed values and create the sampler
    auto randomValues_ = randomValues(shots, 1.0);
    custatevecSamplerDescriptor_t sampler;
    HANDLE_ERROR(custatevecSamplerCreate(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &sampler, shots, &extraWorkspaceSizeInBytes));
    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    // Run the sampling preprocess step.
    HANDLE_ERROR(custatevecSamplerPreprocess(handle, sampler, extraWorkspace,
                                             extraWorkspaceSizeInBytes));

    // Sample!
    custatevecIndex_t bitstrings0[shots];
    HANDLE_ERROR(custatevecSamplerSample(
        handle, sampler, bitstrings0, measuredBits32.data(),
        measuredBits32.size(), randomValues_.data(), shots,
        CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

    if (extraWorkspace) {
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
      extraWorkspace = nullptr;
    }

    std::vector<std::string> sequentialData;

    cudaq::ExecutionResult counts;

    // We've sampled, convert the results to our ExecutionResult counts
    for (int i = 0; i < shots; ++i) {
      auto bitstring = std::bitset<64>(bitstrings0[i])
                           .to_string()
                           .erase(0, 64 - measuredBits.size());
      std::reverse(bitstring.begin(), bitstring.end());
      sequentialData.push_back(bitstring);
      counts.appendResult(bitstring, 1);
    }

    // Compute the expectation value from the counts
    for (auto &kv : counts.counts) {
      auto par = cudaq::sample_result::has_even_parity(kv.first);
      auto p = kv.second / (double)shots;
      if (!par) {
        p = -p;
      }
      expVal += p;
    }

    counts.expectationValue = expVal;
    return counts;
  }

  std::unique_ptr<cudaq::SimulationState> getSimulationState() override {
    flushGateQueue();
    ownsDeviceVector = false;
    return std::make_unique<cudaq::CusvState<ScalarType>>(stateDimension,
                                                          deviceStateVector);
  }

  bool isStateVectorSimulator() const override { return true; }

  std::string name() const override;
  NVQIR_SIMULATOR_CLONE_IMPL(CuStateVecCircuitSimulator<ScalarType>)
};
} // namespace

#ifndef __NVQIR_CUSTATEVEC_TOGGLE_CREATE
template <>
std::string CuStateVecCircuitSimulator<double>::name() const {
  return "custatevec-fp64";
}
/// Register this Simulator with NVQIR.
NVQIR_REGISTER_SIMULATOR(CuStateVecCircuitSimulator<>, custatevec_fp64)
#endif
