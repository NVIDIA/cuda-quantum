#include "hip/hip_runtime.h"
/*************************************************************** -*- C++ -*- ***
 * Copyright (c) 2022 - 2023 NVIDIA Corporation & Affiliates.                  *
 * All rights reserved.                                                        *
 *                                                                             *
 * This source code and the accompanying materials are made available under    *
 * the terms of the Apache License 2.0 which accompanies this distribution.    *
 ******************************************************************************/

#pragma nv_diag_suppress = unsigned_compare_with_zero
#pragma nv_diag_suppress = unrecognized_gcc_pragma

#include "CircuitSimulator.h"
#include "Gates.h"
#include "CuStateVecState.h"

#include "hip/hip_complex.h"
#include "custatevec.h"
#include <bitset>
#include <complex>
#include <iostream>
#include <random>
#include <set>

namespace {

/// @brief Initialize the device state vector to the |0...0> state
/// @param sv
/// @param dim
/// @return
template <typename CudaDataType>
__global__ void initializeDeviceStateVector(CudaDataType *sv, int64_t dim) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i == 0) {
    sv[i].x = 1.0;
    sv[i].y = 0.0;
  } else if (i < dim) {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

/// @brief Kernel to set the first N elements of the state vector sv equal to
/// the
// elements provided by the vector sv2. N is the number of elements to set.
// Size of sv must be greater than size of sv2.
/// @param sv
/// @param sv2
/// @param N
/// @return
template <typename T>
__global__ void setFirstNElements(T *sv, const T *__restrict__ sv2, int64_t N) {
  int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (i < N) {
    sv[i].x = sv2[i].x;
    sv[i].y = sv2[i].y;
  } else {
    sv[i].x = 0.0;
    sv[i].y = 0.0;
  }
}

// template <typename T>
// using ThrustComplex = thrust::complex<T>;

// /// @brief Custom functor for the thrust inner product.
// template <typename T>
// struct AdotConjB
//     : public thrust::binary_function<ThrustComplex<T>, ThrustComplex<T>,
//                                      ThrustComplex<T>> {
//   __host__ __device__ ThrustComplex<T> operator()(ThrustComplex<T> a,
//                                                   ThrustComplex<T> b) {
//     return a * thrust::conj(b);
//   };
// };

// // /// @brief CusvState provides an implementation of `SimulationState` that
// /// encapsulates the state data for the Custatevec Circuit Simulator. It
// /// attempts to keep the simulation data on GPU device and care is taken 
// /// to ensure operations and comparisons with other states operate 
// /// on compatible floating point element types.
// template <typename ScalarType>
// struct CusvState : public cudaq::SimulationState {
// private:
//   /// @brief Size of the state data array on GPU.
//   std::size_t size = 0;

//   /// @brief State device data pointer.
//   void *devicePtr = nullptr;

//   /// @brief Check that we are currently
//   /// using the correct CUDA device, set it
//   /// to the correct one if not
//   void checkAndSetDevice() const {
//     int dev = 0;
//     HANDLE_CUDA_ERROR(hipGetDevice(&dev));
//     auto currentDevice = deviceFromPointer(devicePtr);
//     if (dev != currentDevice)
//       HANDLE_CUDA_ERROR(hipSetDevice(currentDevice));
//   }

//   /// @brief Extract state vector amplitudes from the
//   /// given range.
//   void extractValues(std::complex<ScalarType> *value, std::size_t start,
//                      std::size_t end) const {
//     checkAndSetDevice();
//     HANDLE_CUDA_ERROR(hipMemcpy(
//         value, reinterpret_cast<std::complex<ScalarType> *>(devicePtr) + start,
//         (end - start) * sizeof(std::complex<ScalarType>),
//         hipMemcpyDeviceToHost));
//   }

//   /// @brief Return true if the given pointer is a GPU device pointer
//   bool isDevicePointer(void *ptr) const {
//     hipPointerAttribute_t attributes;
//     HANDLE_CUDA_ERROR(hipPointerGetAttributes(&attributes, ptr));
//     return attributes.type > 1;
//   }

//   /// @brief Given a GPU device pointer, get the CUDA device it is on.
//   std::size_t deviceFromPointer(void *ptr) const {
//     hipPointerAttribute_t attributes;
//     HANDLE_CUDA_ERROR(hipPointerGetAttributes(&attributes, ptr));
//     return attributes.device;
//   }

//   /// @brief Internal utility method for computing overlap from
//   /// validated pointer. This method combines common code from
//   /// the `overlap(T*,size_t)` overloads.
//   template <typename T>
//   double internalOverlapVectorImpl(const std::vector<std::complex<T>> &other) {
//     // Cast our data pointer to be compatible with Thrust.
//     auto *castedDevicePtr = reinterpret_cast<ThrustComplex<T> *>(devicePtr);
//     thrust::device_ptr<ThrustComplex<T>> thrustDevPtrABegin(castedDevicePtr);
//     thrust::device_ptr<ThrustComplex<T>> thrustDevPtrAEnd(castedDevicePtr +
//                                                           size);

//     // Here we explicitly copy the data to the GPU
//     thrust::device_vector<ThrustComplex<T>> otherDevPtr(other);
//     return thrust::inner_product(thrustDevPtrABegin, thrustDevPtrAEnd,
//                                  otherDevPtr.begin(), ThrustComplex<T>(0.0),
//                                  thrust::plus<ThrustComplex<T>>(),
//                                  AdotConjB<T>())
//         .real();
//   }

//   /// @brief Internal utility method for computing overlap from
//   /// validated pointer. This method combines common code from
//   /// the `overlap(T*,size_t)` overloads.
//   template <typename T>
//   double internalOverlapPointerImpl(std::complex<T> *other) {
//     // Cast the data to a Thrust compatible type
//     auto *castedOther = reinterpret_cast<ThrustComplex<T> *>(other);
//     auto *castedDevicePtr = reinterpret_cast<ThrustComplex<T> *>(devicePtr);
//     thrust::device_ptr<ThrustComplex<T>> thrustDevPtrABegin(castedDevicePtr);
//     thrust::device_ptr<ThrustComplex<T>> thrustDevPtrAEnd(castedDevicePtr +
//                                                           size);

//     // Check that the other pointer is on GPU device
//     if (!isDevicePointer(other)) {
//       // here we have to copy the data
//       thrust::device_vector<ThrustComplex<T>> otherDevPtr(castedOther,
//                                                           castedOther + size);
//       return thrust::inner_product(thrustDevPtrABegin, thrustDevPtrAEnd,
//                                    otherDevPtr.begin(), ThrustComplex<T>(0.0),
//                                    thrust::plus<ThrustComplex<T>>(),
//                                    AdotConjB<T>())
//           .real();
//     }

//     // We have two device pointers, make sure they are on the same CUDA device
//     if (deviceFromPointer(devicePtr) != deviceFromPointer(other))
//       throw std::runtime_error("[custatevec-state] overlap requested for "
//                                "device pointers on separate GPU devices.");

//     // Compute the overlap
//     thrust::device_ptr<ThrustComplex<T>> thrustDevPtrBBegin(castedOther);
//     return thrust::inner_product(thrustDevPtrABegin, thrustDevPtrAEnd,
//                                  &thrustDevPtrBBegin[0], ThrustComplex<T>(0.0),
//                                  thrust::plus<ThrustComplex<T>>(),
//                                  AdotConjB<T>())
//         .real();
//   }

// public:
//   CusvState(std::size_t s, void *ptr) : size(s), devicePtr(ptr) {}

//   /// @brief Return the number of qubits this state models
//   std::size_t getNumQubits() const override { return std::log2(size); }

//   /// @brief Return the shape of the data.
//   std::vector<std::size_t> getDataShape() const override { return {size}; }

//   /// @brief Compute the overlap of this state with the provided one.
//   /// If the other state is not on GPU device, this function will
//   /// copy the data from host.
//   double overlap(const cudaq::SimulationState &other) override {
//     if (getDataShape() != other.getDataShape())
//       throw std::runtime_error("[custatevec-state] overlap error - other state "
//                                "dimension not equal to this state dimension.");

//     if (other.getPrecision() != getPrecision()) {
//       throw std::runtime_error(
//           "[custatevec-state] overlap error - precision mismatch.");
//     }

//     // Cast our data pointer to be compatible with Thrust.
//     auto *castedDevicePtr =
//         reinterpret_cast<ThrustComplex<ScalarType> *>(devicePtr);
//     thrust::device_ptr<ThrustComplex<ScalarType>> thrustDevPtrABegin(
//         castedDevicePtr);
//     thrust::device_ptr<ThrustComplex<ScalarType>> thrustDevPtrAEnd(
//         castedDevicePtr + size);

//     // Make sure other is on GPU device already
//     if (isDevicePointer(other.ptr())) {
//       if (deviceFromPointer(devicePtr) != deviceFromPointer(other.ptr()))
//         throw std::runtime_error(
//             "overlap requested for device pointers on separate GPU devices.");
//       // other is a device pointer
//       thrust::device_ptr<ThrustComplex<ScalarType>> thrustDevPtrBBegin(
//           reinterpret_cast<ThrustComplex<ScalarType> *>(other.ptr()));
//       return thrust::inner_product(thrustDevPtrABegin, thrustDevPtrAEnd,
//                                    thrustDevPtrBBegin,
//                                    ThrustComplex<ScalarType>(0.0),
//                                    thrust::plus<ThrustComplex<ScalarType>>(),
//                                    AdotConjB<ScalarType>())
//           .real();
//     }

//     // If we reach here, then we have to copy the data from host.
//     cudaq::info(
//         "[custatevec-state] overlap computation requested with a state that is "
//         "in host memory. Host data will be copied to GPU.");

//     // Cast the other pointer to be compatible with Thrust.
//     auto *castedOtherPtr =
//         reinterpret_cast<std::complex<ScalarType> *>(other.ptr());
//     std::vector<std::complex<ScalarType>> dataAsVec(castedOtherPtr,
//                                                     castedOtherPtr + size);
//     thrust::device_vector<ThrustComplex<ScalarType>> otherDevPtr(dataAsVec);
//     return thrust::inner_product(thrustDevPtrABegin, thrustDevPtrAEnd,
//                                  otherDevPtr.begin(),
//                                  ThrustComplex<ScalarType>(0.0),
//                                  thrust::plus<ThrustComplex<ScalarType>>(),
//                                  AdotConjB<ScalarType>())
//         .real();
//   }

//   /// @brief Compute the overlap of this state with the data provided as a
//   /// `std::vector<double>`. If this device state is not FP64, throw an
//   /// exception. This overload requires an explicit copy from host memory.
//   double overlap(const std::vector<cudaq::complex128> &other) override {
//     // We must use compatible element types
//     if constexpr (std::is_same_v<ScalarType, float>) {
//       throw std::runtime_error("simulation precision is FP32 but overlap "
//                                "requested with FP64 state data.");
//     }

//     // Beyond here, ScalarType can only be == double

//     // Check that our shapes are correct
//     if (getDataShape()[0] != other.size())
//       throw std::runtime_error("[custatevec-state] overlap error - other state "
//                                "dimension not equal to this state dimension.");

//     return internalOverlapVectorImpl<double>(other);
//   }

//   /// @brief Compute the overlap of this state with the data provided as a
//   /// `std::vector<float>`. If this device state is not FP32, throw an
//   /// exception. This overload requires an explicit copy from host memory.
//   double overlap(const std::vector<cudaq::complex64> &other) override {
//     if constexpr (std::is_same_v<ScalarType, double>) {
//       throw std::runtime_error("simulation precision is FP64 but overlap "
//                                "requested with FP32 state data.");
//     }

//     // Beyond here, ScalarType can only be == float

//     // Check that are shapes are correct
//     if (getDataShape()[0] != other.size())
//       throw std::runtime_error("[custatevec-state] overlap error - other state "
//                                "dimension not equal to this state dimension.");

//     return internalOverlapVectorImpl<float>(other);
//   }

//   /// @brief Compute the overlap of this state with the data provided as a raw
//   /// pointer. This overload will check if this pointer corresponds to a device
//   /// pointer. It will copy the data from host to device if necessary.
//   double overlap(cudaq::complex128 *other, std::size_t numElements) override {
//     // Must have the correct precision
//     if constexpr (std::is_same_v<ScalarType, float>) {
//       throw std::runtime_error("simulation precision is FP32 but overlap "
//                                "requested with FP64 state data.");
//     }

//     // Must have the correct number of elements.
//     if (numElements != size)
//       throw std::runtime_error("[custatevec-state] overlap with pointer, "
//                                "invalid number of elements specified.");

//     return internalOverlapPointerImpl<double>(other);
//   }

//   double overlap(cudaq::complex64 *other, std::size_t numElements) override {
//     // Must have the correct precision
//     if constexpr (std::is_same_v<ScalarType, double>) {
//       throw std::runtime_error("simulation precision is FP64 but overlap "
//                                "requested with FP32 state data.");
//     }

//     // Must have the correct number of elements.
//     if (numElements != size)
//       throw std::runtime_error("[custatevec-state] overlap with pointer, "
//                                "invalid number of elements specified.");

//     return internalOverlapPointerImpl<float>(other);
//   }

//   /// @brief Return the vector element at the given index.
//   cudaq::complex128 vectorElement(std::size_t idx) override {
//     std::complex<ScalarType> value;
//     extractValues(&value, idx, idx + 1);
//     return {value.real(), value.imag()};
//   }

//   /// @brief Dump the state to the given output stream
//   void dump(std::ostream &os) const override {
//     // get state data from device to print
//     std::vector<std::complex<ScalarType>> tmp(size);
//     HANDLE_CUDA_ERROR(hipMemcpy(tmp.data(), devicePtr,
//                                  size * sizeof(std::complex<ScalarType>),
//                                  hipMemcpyDeviceToHost));
//     for (auto &t : tmp)
//       os << t << "\n";
//   }

//   /// @brief This state is GPU device data, always return true.
//   bool isDeviceData() const override { return true; }

//   /// @brief Copy the state device data to the user-provided host data pointer.
//   void toHost(cudaq::complex128 *userData,
//               std::size_t numElements) const override {
//     // Must have the correct precision
//     if constexpr (std::is_same_v<ScalarType, float>) {
//       throw std::runtime_error("simulation precision is FP32 but overlap "
//                                "requested with FP64 state data.");
//     }
//     // Must have the correct number of elements.
//     if (numElements != size)
//       throw std::runtime_error("[custatevec-state] provided toHost pointer has "
//                                "invalid number of elements specified.");

//     extractValues(reinterpret_cast<std::complex<ScalarType> *>(userData), 0,
//                   size);
//     return;
//   }

//   /// @brief Copy the state device data to the user-provided host data pointer.
//   void toHost(cudaq::complex64 *userData,
//               std::size_t numElements) const override {
//     // Must have the correct precision
//     if constexpr (std::is_same_v<ScalarType, float>) {
//       throw std::runtime_error("simulation precision is FP32 but overlap "
//                                "requested with FP64 state data.");
//     }
//     // Must have the correct number of elements.
//     if (numElements != size)
//       throw std::runtime_error("[custatevec-state] provided toHost pointer has "
//                                "invalid number of elements specified.");

//     extractValues(reinterpret_cast<std::complex<ScalarType> *>(userData), 0,
//                   size);
//     return;
//   }

//   /// @brief Return the raw pointer to the device data.
//   void *ptr() const override { return devicePtr; }

//   /// @brief Return the precision of the state data elements.
//   precision getPrecision() const override {
//     if constexpr (std::is_same_v<ScalarType, float>)
//       return cudaq::SimulationState::precision::fp32;
//     return cudaq::SimulationState::precision::fp64;
//   }

//   /// @brief Free the device data.
//   void destroyState() override {
//     cudaq::info("custatevec-state destroying state vector handle.");
//     HANDLE_CUDA_ERROR(hipFree(devicePtr));
//   }
// };

/// @brief The CuStateVecCircuitSimulator implements the CircuitSimulator
/// base class to provide a simulator that delegates to the NVIDIA CuStateVec
/// GPU-accelerated library.
template <typename ScalarType = double>
class CuStateVecCircuitSimulator
    : public nvqir::CircuitSimulatorBase<ScalarType> {
protected:
  // This type by default uses FP64
  using DataType = std::complex<ScalarType>;
  using DataVector = std::vector<DataType>;
  using CudaDataType = std::conditional_t<std::is_same_v<ScalarType, float>,
                                          hipFloatComplex, hipDoubleComplex>;

  using nvqir::CircuitSimulatorBase<ScalarType>::tracker;
  using nvqir::CircuitSimulatorBase<ScalarType>::nQubitsAllocated;
  using nvqir::CircuitSimulatorBase<ScalarType>::stateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::calculateStateDim;
  using nvqir::CircuitSimulatorBase<ScalarType>::executionContext;
  using nvqir::CircuitSimulatorBase<ScalarType>::gateToString;
  using nvqir::CircuitSimulatorBase<ScalarType>::x;
  using nvqir::CircuitSimulatorBase<ScalarType>::flushGateQueue;
  using nvqir::CircuitSimulatorBase<ScalarType>::previousStateDimension;
  using nvqir::CircuitSimulatorBase<ScalarType>::shouldObserveFromSampling;

  /// @brief The statevector that cuStateVec manipulates on the GPU
  void *deviceStateVector = nullptr;

  /// @brief The cuStateVec handle
  custatevecHandle_t handle;

  /// @brief Pointer to potentially needed extra memory
  void *extraWorkspace = nullptr;

  /// @brief The size of the extra workspace
  size_t extraWorkspaceSizeInBytes = 0;

  custatevecComputeType_t cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
  hipDataType cuStateVecCudaDataType = HIP_C_64F;
  std::random_device randomDevice;
  std::mt19937 randomEngine;
  bool ownsDeviceVector = true;

  /// @brief Generate a vector of random values
  std::vector<double> randomValues(uint64_t num_samples, double max_value) {
    std::vector<double> rs;
    rs.reserve(num_samples);
    std::uniform_real_distribution<double> distr(0.0, max_value);
    for (uint64_t i = 0; i < num_samples; ++i) {
      rs.emplace_back(distr(randomEngine));
    }
    std::sort(rs.begin(), rs.end());
    return rs;
  }

  /// @brief Convert the pauli rotation gate name to a CUSTATEVEC_PAULI Type
  /// @param type
  /// @return
  custatevecPauli_t pauliStringToEnum(const std::string_view type) {
    if (type == "rx") {
      return CUSTATEVEC_PAULI_X;
    } else if (type == "ry") {
      return CUSTATEVEC_PAULI_Y;
    } else if (type == "rz") {
      return CUSTATEVEC_PAULI_Z;
    }
    printf("Error, should not be here with pauli.\n");
    exit(1);
  }

  /// @brief Apply the matrix to the state vector on the GPU
  /// @param matrix The matrix data as a 1-d array, row-major
  /// @param controls Possible control qubits, can be empty
  /// @param targets Target qubits
  void applyGateMatrix(const DataVector &matrix,
                       const std::vector<int> &controls,
                       const std::vector<int> &targets) {
    HANDLE_ERROR(custatevecApplyMatrixGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nQubitsAllocated, matrix.data(),
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.size(),
        controls.size(), cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0)
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

    auto localNQubitsAllocated =
        stateDimension > 0 ? std::log2(stateDimension) : 0;

    // apply gate
    HANDLE_ERROR(custatevecApplyMatrix(
        handle, deviceStateVector, cuStateVecCudaDataType,
        localNQubitsAllocated, matrix.data(), cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets.data(), targets.size(),
        controls.empty() ? nullptr : controls.data(), nullptr, controls.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
  }

  /// @brief Utility function for applying one-target-qubit rotation operations
  template <typename RotationGateT>
  void oneQubitOneParamApply(const double angle,
                             const std::vector<std::size_t> &controls,
                             const std::size_t qubitIdx) {
    RotationGateT gate;
    std::vector<int> controls32;
    for (auto c : controls)
      controls32.push_back((int)c);
    custatevecPauli_t pauli[] = {pauliStringToEnum(gate.name())};
    int targets[] = {(int)qubitIdx};
    custatevecApplyPauliRotation(handle, deviceStateVector,
                                 cuStateVecCudaDataType, nQubitsAllocated,
                                 -0.5 * angle, pauli, targets, 1,
                                 controls32.data(), nullptr, controls32.size());
  }

  /// @brief Increase the state size by the given number of qubits.
  void addQubitsToState(std::size_t count) override {
    if (count == 0)
      return;

    int dev;
    HANDLE_CUDA_ERROR(hipGetDevice(&dev));
    cudaq::info("GPU {} Allocating new qubit array of size {}.", dev, count);

    if (!deviceStateVector) {
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
      HANDLE_ERROR(custatevecCreate(&handle));
    } else {
      // Allocate new state..
      void *newDeviceStateVector;
      HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      setFirstNElements<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(newDeviceStateVector),
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          previousStateDimension);
      hipFree(deviceStateVector);
      deviceStateVector = newDeviceStateVector;
    }
  }

  /// @brief Increase the state size by one qubit.
  void addQubitToState() override {
    // Update the state vector
    if (!deviceStateVector) {
      HANDLE_CUDA_ERROR(hipMalloc((void **)&deviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
      HANDLE_ERROR(custatevecCreate(&handle));
    } else {
      // Allocate new state..
      void *newDeviceStateVector;
      HANDLE_CUDA_ERROR(hipMalloc((void **)&newDeviceStateVector,
                                   stateDimension * sizeof(CudaDataType)));
      constexpr int32_t threads_per_block = 256;
      uint32_t n_blocks =
          (stateDimension + threads_per_block - 1) / threads_per_block;
      setFirstNElements<<<n_blocks, threads_per_block>>>(
          reinterpret_cast<CudaDataType *>(newDeviceStateVector),
          reinterpret_cast<CudaDataType *>(deviceStateVector),
          previousStateDimension);
      hipFree(deviceStateVector);
      deviceStateVector = newDeviceStateVector;
    }
  }

  /// @brief Reset the qubit state.
  void deallocateStateImpl() override {
    if (deviceStateVector) {
      HANDLE_ERROR(custatevecDestroy(handle));
      if (ownsDeviceVector)
        HANDLE_CUDA_ERROR(hipFree(deviceStateVector));
    }
    if (extraWorkspaceSizeInBytes)
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
    deviceStateVector = nullptr;
    extraWorkspaceSizeInBytes = 0;
  }

  /// @brief Apply the given GateApplicationTask
  void applyGate(const typename nvqir::CircuitSimulatorBase<
                 ScalarType>::GateApplicationTask &task) override {
    std::vector<int> controls, targets;
    std::transform(task.controls.begin(), task.controls.end(),
                   std::back_inserter(controls),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    std::transform(task.targets.begin(), task.targets.end(),
                   std::back_inserter(targets),
                   [](std::size_t idx) { return static_cast<int>(idx); });
    // If we have no parameters, just apply the matrix.
    if (task.parameters.empty()) {
      applyGateMatrix(task.matrix, controls, targets);
      return;
    }

    // If we have parameters, it may be more efficient to
    // compute with custatevecApplyPauliRotation
    if (task.operationName == "rx") {
      oneQubitOneParamApply<nvqir::rx<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "ry") {
      oneQubitOneParamApply<nvqir::ry<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else if (task.operationName == "rz") {
      oneQubitOneParamApply<nvqir::rz<ScalarType>>(
          task.parameters[0], task.controls, task.targets[0]);
    } else {
      // Fallback to just applying the gate.
      applyGateMatrix(task.matrix, controls, targets);
    }
  }

  /// @brief Set the state back to the |0> state on the
  /// current number of qubits
  void setToZeroState() override {
    constexpr int32_t threads_per_block = 256;
    uint32_t n_blocks =
        (stateDimension + threads_per_block - 1) / threads_per_block;
    initializeDeviceStateVector<<<n_blocks, threads_per_block>>>(
        reinterpret_cast<CudaDataType *>(deviceStateVector), stateDimension);
  }

public:
  /// @brief The constructor
  CuStateVecCircuitSimulator() {
    if constexpr (std::is_same_v<ScalarType, float>) {
      cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
      cuStateVecCudaDataType = HIP_C_32F;
    }

    hipFree(0);
    randomEngine = std::mt19937(randomDevice());
  }

  /// The destructor
  virtual ~CuStateVecCircuitSimulator() = default;

  void setRandomSeed(std::size_t randomSeed) override {
    randomEngine = std::mt19937(randomSeed);
  }

  /// @brief Measure operation
  /// @param qubitIdx
  /// @return
  bool measureQubit(const std::size_t qubitIdx) override {
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    cudaq::info("Measured qubit {} -> {}", qubitIdx, parity);
    return parity == 1 ? true : false;
  }

  /// @brief Reset the qubit
  /// @param qubitIdx
  void resetQubit(const std::size_t qubitIdx) override {
    flushGateQueue();
    const int basisBits[] = {(int)qubitIdx};
    int parity;
    double rand = randomValues(1, 1.0)[0];
    HANDLE_ERROR(custatevecMeasureOnZBasis(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &parity, basisBits, /*N Bits*/ 1, rand,
        CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO));
    if (parity) {
      x(qubitIdx);
    }
  }

  /// @brief Override base class functionality for a general Pauli
  /// rotation to delegate to the performant custatevecApplyPauliRotation.
  void applyExpPauli(double theta, const std::vector<std::size_t> &controlIds,
                     const std::vector<std::size_t> &qubits,
                     const cudaq::spin_op &op) override {
    flushGateQueue();
    cudaq::info(" [cusv decomposing] exp_pauli({}, {})", theta,
                op.to_string(false));
    std::vector<int> controls, targets;
    for (const auto &bit : controlIds)
      controls.emplace_back(static_cast<int>(bit));
    std::vector<custatevecPauli_t> paulis;
    op.for_each_pauli([&](cudaq::pauli p, std::size_t i) {
      if (p == cudaq::pauli::I)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_I);
      else if (p == cudaq::pauli::X)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_X);
      else if (p == cudaq::pauli::Y)
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Y);
      else
        paulis.push_back(custatevecPauli_t::CUSTATEVEC_PAULI_Z);

      targets.push_back(qubits[i]);
    });

    HANDLE_ERROR(custatevecApplyPauliRotation(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        theta, paulis.data(), targets.data(), targets.size(), controls.data(),
        nullptr, controls.size()));
  }

  /// @brief Compute the operator expectation value, with respect to
  /// the current state vector, directly on GPU with the
  /// given the operator matrix and target qubit indices.
  auto getExpectationFromOperatorMatrix(const std::complex<double> *matrix,
                                        const std::vector<std::size_t> &tgts) {
    void *extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // Convert the size_t tgts into ints
    std::vector<int> tgtsInt(tgts.size());
    std::transform(tgts.begin(), tgts.end(), tgtsInt.begin(),
                   [&](std::size_t x) { return static_cast<int>(x); });
    // our bit ordering is reversed.
    size_t nIndexBits = nQubitsAllocated;

    // check the size of external workspace
    HANDLE_ERROR(custatevecComputeExpectationGetWorkspaceSize(
        handle, cuStateVecCudaDataType, nIndexBits, matrix,
        cuStateVecCudaDataType, CUSTATEVEC_MATRIX_LAYOUT_ROW, tgts.size(),
        cuStateVecComputeType, &extraWorkspaceSizeInBytes));

    if (extraWorkspaceSizeInBytes > 0) {
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));
    }

    double expect;

    // compute expectation
    HANDLE_ERROR(custatevecComputeExpectation(
        handle, deviceStateVector, cuStateVecCudaDataType, nIndexBits, &expect,
        HIP_R_64F, nullptr, matrix, cuStateVecCudaDataType,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, tgtsInt.data(), tgts.size(),
        cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
    if (extraWorkspaceSizeInBytes)
      HANDLE_CUDA_ERROR(hipFree(extraWorkspace));

    return expect;
  }

  /// @brief We can compute Observe from the matrix for a
  /// reasonable number of qubits, otherwise we should compute it
  /// via sampling
  bool canHandleObserve() override {
    // Do not compute <H> from matrix if shots based sampling requested
    if (executionContext &&
        executionContext->shots != static_cast<std::size_t>(-1)) {
      return false;
    }

    /// Seems that FP32 is faster with
    /// custatevecComputeExpectationsOnPauliBasis
    if constexpr (std::is_same_v<ScalarType, float>) {
      return false;
    }

    return !shouldObserveFromSampling();
  }

  /// @brief Compute the expected value from the observable matrix.
  cudaq::ExecutionResult observe(const cudaq::spin_op &op) override {

    flushGateQueue();

    // The op is on the following target bits.
    std::set<std::size_t> targets;
    op.for_each_term([&](cudaq::spin_op &term) {
      term.for_each_pauli(
          [&](cudaq::pauli p, std::size_t idx) { targets.insert(idx); });
    });

    std::vector<std::size_t> targetsVec(targets.begin(), targets.end());

    // Get the matrix
    auto matrix = op.to_matrix();
    /// Compute the expectation value.
    auto ee = getExpectationFromOperatorMatrix(matrix.data(), targetsVec);
    return cudaq::ExecutionResult({}, ee);
  }

  /// @brief Sample the multi-qubit state.
  cudaq::ExecutionResult sample(const std::vector<std::size_t> &measuredBits,
                                const int shots) override {
    double expVal = 0.0;
    // cudaq::CountsDictionary counts;
    std::vector<custatevecPauli_t> z_pauli;
    std::vector<int> measuredBits32;
    for (auto m : measuredBits) {
      measuredBits32.push_back(m);
      z_pauli.push_back(CUSTATEVEC_PAULI_Z);
    }

    if (shots < 1) {
      // Just compute the expected value on <Z...Z>
      const uint32_t nBasisBitsArray[] = {(uint32_t)measuredBits.size()};
      const int *basisBitsArray[] = {measuredBits32.data()};
      const custatevecPauli_t *pauliArray[] = {z_pauli.data()};
      double expectationValues[1];
      HANDLE_ERROR(custatevecComputeExpectationsOnPauliBasis(
          handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
          expectationValues, pauliArray, 1, basisBitsArray, nBasisBitsArray));
      expVal = expectationValues[0];
      cudaq::info("Computed expectation value = {}", expVal);
      return cudaq::ExecutionResult{expVal};
    }

    // Grab some random seed values and create the sampler
    auto randomValues_ = randomValues(shots, 1.0);
    custatevecSamplerDescriptor_t sampler;
    HANDLE_ERROR(custatevecSamplerCreate(
        handle, deviceStateVector, cuStateVecCudaDataType, nQubitsAllocated,
        &sampler, shots, &extraWorkspaceSizeInBytes));
    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0) {
      HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));
    }

    // Run the sampling preprocess step.
    HANDLE_ERROR(custatevecSamplerPreprocess(handle, sampler, extraWorkspace,
                                             extraWorkspaceSizeInBytes));

    // Sample!
    custatevecIndex_t bitstrings0[shots];
    HANDLE_ERROR(custatevecSamplerSample(
        handle, sampler, bitstrings0, measuredBits32.data(),
        measuredBits32.size(), randomValues_.data(), shots,
        CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

    std::vector<std::string> sequentialData;

    cudaq::ExecutionResult counts;

    // We've sampled, convert the results to our ExecutionResult counts
    for (int i = 0; i < shots; ++i) {
      auto bitstring = std::bitset<64>(bitstrings0[i])
                           .to_string()
                           .erase(0, 64 - measuredBits.size());
      std::reverse(bitstring.begin(), bitstring.end());
      sequentialData.push_back(bitstring);
      counts.appendResult(bitstring, 1);
    }

    // Compute the expectation value from the counts
    for (auto &kv : counts.counts) {
      auto par = cudaq::sample_result::has_even_parity(kv.first);
      auto p = kv.second / (double)shots;
      if (!par) {
        p = -p;
      }
      expVal += p;
    }

    counts.expectationValue = expVal;
    return counts;
  }

  std::unique_ptr<cudaq::SimulationState> getSimulationState() override {
    flushGateQueue();
    ownsDeviceVector = false;
    return std::make_unique<cudaq::CusvState<ScalarType>>(stateDimension,
                                                   deviceStateVector);
  }

  std::string name() const override;
  NVQIR_SIMULATOR_CLONE_IMPL(CuStateVecCircuitSimulator<ScalarType>)
};
} // namespace

#ifndef __NVQIR_CUSTATEVEC_TOGGLE_CREATE
template <>
std::string CuStateVecCircuitSimulator<double>::name() const {
  return "custatevec-fp64";
}
/// Register this Simulator with NVQIR.
NVQIR_REGISTER_SIMULATOR(CuStateVecCircuitSimulator<>, custatevec_fp64)
#endif
